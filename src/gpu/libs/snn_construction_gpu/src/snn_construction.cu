#include "hip/hip_runtime.h"
#include <snn_construction.cuh>


void __global__ fill_N_G_group_id_and_G_neuron_count_per_type_(
	const int N,
    const int G,
	const float* N_pos, 
	float N_pos_shape_x, float N_pos_shape_y, float N_pos_shape_z,
    const int N_pos_n_cols,
	int* N_G,
    const int N_G_n_cols,
	const int N_G_neuron_type_col,
    const int N_G_group_id_col,

	const float G_shape_x,
	const float G_shape_y,
	const float G_shape_z,
	const float min_G_shape,

	int* G_neuron_counts  // NOLINT(readability-non-const-parameter)
)
{
	const int n = blockIdx.x * blockDim.x + threadIdx.x;  // NOLINT(bugprone-narrowing-conversions)
	
	if (n < N)
	{
		const int x = __float2int_rd(((N_pos[n * N_pos_n_cols] / N_pos_shape_x) * min_G_shape));  
		const int y = __float2int_rd(((N_pos[n * N_pos_n_cols + 1] / N_pos_shape_y) * min_G_shape));
		const int z = __float2int_rd(((N_pos[n * N_pos_n_cols + 2] / N_pos_shape_z) * min_G_shape));

		const int group = x + G_shape_x * y + G_shape_x * G_shape_y * z;
        
		const int row_idx = n * N_G_n_cols;

		// assign neuron to location-based group 
		N_G[row_idx + N_G_group_id_col] = group;
		
		// count group: row <> neuron type (1 or 2), column <> group id
		atomicAdd(&G_neuron_counts[group + G * (N_G[row_idx + N_G_neuron_type_col] - 1)], 1);	
	}
}


void fill_N_G_group_id_and_G_neuron_count_per_type(
    const int N, 
    const int G, 
    const float* N_pos,
	const int N_pos_shape_x, const int N_pos_shape_y, const int N_pos_shape_z,
    int* N_G,
	int* G_neuron_counts,
    const int G_shape_x, const int G_shape_y, const int G_shape_z,
	const int N_pos_n_cols,
	const int N_G_n_cols,
	const int N_G_neuron_type_col,
	const int N_G_group_id_col
)
{	
	// Assign location-based group ids to neurons w.r.t. their positions.

	// N: 				number of neurons
	// G: 				number of location-based groups
	// N_pos: device 	pointer to the position array
	// N_pos_n_cols: 	number of columns of N_pos
	// N_G: 			device pointer to the neuron-group-info array
	// N_G_n_cols:	 	number of columns of N_G
	// N_G_g_id_col:	column in which to write the group-id
	// G_shape_*:		number of location-based groups along the *-axis
	// 
	//
	// Example:
	// 
	//	G = 8
	//	N_pos_n_cols = 3
	//	N_pos = [[0,0,0], 
	//			 [1,1,0]]
	//		
	//	N_G_group_id_col = 1
	//
	// 	N_G = 	[[0,0],    -> 	N_G = 	[[0,0]
	//			 [1,0]]					 [1,2]]
	//	
	//	G_neuron_counts = 	[[0,0,0],	-> 	[[1,0,0],
	//						 [0,0,0]]		 [0,0,1]]
	//	


	hipDeviceSynchronize();
	LaunchParameters launch(N, (void *)fill_N_G_group_id_and_G_neuron_count_per_type_); 
	
	int min_G_shape = std::min(std::min(G_shape_x, G_shape_y), G_shape_z);

	fill_N_G_group_id_and_G_neuron_count_per_type_ KERNEL_ARGS2(launch.grid3, launch.block3) (
		N,
        G,
		N_pos,
		static_cast<float>(N_pos_shape_x), static_cast<float>(N_pos_shape_y), static_cast<float>(N_pos_shape_z),
        N_pos_n_cols,
		N_G,
		N_G_n_cols,
		N_G_neuron_type_col,
		N_G_group_id_col,
		// LG_neuron_counts.dp,
		static_cast<float>(G_shape_x), static_cast<float>(G_shape_y), static_cast<float>(G_shape_z),
		static_cast<float>(min_G_shape),
		G_neuron_counts
	);
	
	hipDeviceSynchronize();
	printf("\n");
}


__device__ float sigmoidal_connection_probability(
	const float delay,
	const float max_delay,
	const float alpha = 1.f,
	const float beta = 1.f,
	const float gamma = 0.125f
)
{
	const float inv_max_delay = (1.f / max_delay);
	const float normalized_delay = delay * inv_max_delay;
	
	const float sigmoid = 0.5f - (1 / (1 + expf(-(alpha * delay - 1.f))));
	const float offset = inv_max_delay * (0.5f + gamma - gamma * powf(normalized_delay, 2.f));

	return fminf(1.f, beta * (inv_max_delay * sigmoid + offset));
}


__global__ void fill_G_neuron_count_per_delay_(
		const float fS,
		const int D,
		const float fD,
		const int G,
		const int* G_delay_distance,
		int* G_neuron_counts
)
{
	// connection probabilities from inh -> exc
	// connection probabilities from exc -> (inh & exc)

	const int g = threadIdx.x + blockDim.x * blockIdx.x;

	if (g < G)
	{
		const int g_row = g * G;
		
		int delay = 0;
		int count_inh = 0;
		int count_exc = 0;

		const int ioffs_inh = 2 * G + g;
		const int ioffs_exc = (2 + D) * G + g;
	
		for (int h = 0; h < G; h++)
		{

			delay = G_delay_distance[g_row + h];

			count_inh = G_neuron_counts[h];
			count_exc = G_neuron_counts[h + G];

			atomicAdd(&G_neuron_counts[delay * G + ioffs_inh], count_inh);
			atomicAdd(&G_neuron_counts[delay * G + ioffs_exc], count_exc);
		}
	}
}


void fill_G_neuron_count_per_delay(
	const int S,
	const int D,
	const int G,
	const int* G_delay_distance,
	int* G_neuron_counts
)
{	
	hipDeviceSynchronize();
	LaunchParameters launch(G, (void *)fill_G_neuron_count_per_delay_); 

	fill_G_neuron_count_per_delay_ KERNEL_ARGS2(launch.grid3, launch.block3)(
		static_cast<float>(S),
		D,
		static_cast<float>(D),
		G,
		G_delay_distance,
		G_neuron_counts
	);
	
	hipDeviceSynchronize();
	printf("\n");
}


__device__ void expected_syn_count(
	const float fD,
	const int D, 
	const int G, 
	const int* G_neuron_counts,
	const int ioffs_inh,
	const int ioffs_exc,

	const float alpha_inh, const float beta_inh, const float gamma_inh,
	const float alpha_exc, const float beta_exc, const float gamma_exc,
	
	float* exp_cnt_inh, float* exp_cnt_exc,
	const int group,
	const bool verbose = 1,
	const int print_group = 1
)
{
	*exp_cnt_inh = 0;
	*exp_cnt_exc = 0;

	
	for (int delay = 0; delay < D; delay++)
	{
		const int idx = (delay)*G;
		// # inh targets (exc)
		const float n_inh_targets = __int2float_rn(G_neuron_counts[ioffs_exc + idx]);
		// # exc targets (inh & exc)
		float n_exc_targets = n_inh_targets + __int2float_rn(G_neuron_counts[ioffs_inh + idx]);

		if ((delay == 0) && (G_neuron_counts[ioffs_inh - G] > 0))
		{
			// only exc neurons will have a technical probability > 0 to form an autapse
			n_exc_targets-= 1.f;
		}

		const float fdelay = __int2float_rn(delay);

		const float prob_inh = sigmoidal_connection_probability(fdelay, fD, alpha_inh, beta_inh, gamma_inh);
		const float prob_exc = sigmoidal_connection_probability(fdelay, fD, alpha_exc, beta_exc, gamma_exc);
		if (n_inh_targets > 0){
			*exp_cnt_inh += roundf(n_inh_targets * prob_inh + .5);
		}
		if (n_exc_targets > 0){
			*exp_cnt_exc += roundf(n_exc_targets * prob_exc + .5);
		}
		if ((verbose) && (group == print_group)){
			printf("\ninh expected_syncount = %f (++ %f)", *exp_cnt_inh, n_inh_targets * prob_inh);
			printf("\n(exc) a=%f,b=%f,g=%f", alpha_exc, beta_exc, gamma_exc);
			printf("\nexc expected_syncount = %f (++ %f)", *exp_cnt_exc, roundf(n_exc_targets * prob_exc + .5));
		}

	}
}

__device__ void prob_improvement(
	int* mode,
	float* alpha,
	float* beta,
	float* gamma,
	const float expected_count,
	const float error,
	const float fS,
	const float fD,
	const float alpha_delta,
	const float beta_delta,
	const float gamma_delta
	// const int group
)
{
	if (*mode == 0)
	{
		// if (group == 0) printf("\n(%d) beta=%f", group, *beta);
		// if (group == 0) printf("\n(%d) beta_delta=%f", group, beta_delta);
		*beta = fminf(*beta * fmaxf(fminf(fS / (expected_count), 1.f + beta_delta), 1.f - beta_delta), fD * (1- *gamma));
		*mode = 1;
		// if (group == 0) printf("\n(%d) beta=%f", group, *beta);
	}
	else if (*mode == 1)
	{
		// if (group == 0) printf("\n(%d) alpha=%f", group, *alpha);
		*alpha = fmaxf(*alpha + fmaxf(fminf( ( expected_count - fS) / fS, alpha_delta), -alpha_delta),
			0.05f);
		*mode = 0;
		// if (group == 0) printf("\n(%d) alpha=%f", group, *alpha);
	}

	if (error > (fS * 0.1f))
	{
		// if (group == 0) printf("\n(%d) gamma=%f", group, *gamma);
		*gamma = fminf(*gamma * fmaxf(fminf(fS / (expected_count), 1.f + gamma_delta), 1.f - gamma_delta), .3f);
		// if (group == 0) printf("\n(%d) gamma=%f", group, *gamma);
	}
	
}


__device__ int roughly_optimize_connection_probabilites(
	const float fS,
	const float fD,
	const int D,
	const int G,
	const int* G_neuron_counts, 
	const int ioffs_inh, const int ioffs_exc,
	float* p_alpha_inh, float* p_beta_inh, float* p_gamma_inh,
	float* p_alpha_exc, float* p_beta_exc, float* p_gamma_exc,
	const float alpha_delta, const float beta_delta, const float gamma_delta,
	const int group,
	const bool verbose = 1,
	const int print_group = 1
){
	
	int j = 0;

	float exp_cnt_inh = 0.f;
	float exp_cnt_exc = 0.f;
		
	int mode_inh = 0;
	int mode_exc = 0;

	float error_inh = fS;
	float error_exc = fS;
	const float p = (1. / fS);


	while (((error_inh > p) || (error_exc > p)) && (j < 300))
	{
		expected_syn_count(
			fD, 
			D, 
			G, 
			G_neuron_counts,
			ioffs_inh, ioffs_exc,
			*p_alpha_inh, *p_beta_inh, *p_gamma_inh,
			*p_alpha_exc, *p_beta_exc, *p_gamma_exc,
			&exp_cnt_inh, &exp_cnt_exc,
			group,
			verbose, print_group
		);

		error_inh = fabsf(exp_cnt_inh - fS);
		error_exc = fabsf(exp_cnt_exc - fS);
		
		j++;
		
		if ((error_inh > p))
		{
			prob_improvement(&mode_inh,
				p_alpha_inh, p_beta_inh, p_gamma_inh,
			 	exp_cnt_inh, error_inh,
			 	fS, fD,
			 	alpha_delta, beta_delta, gamma_delta
				//, group 
			);
		}
		if ((error_exc > p))
		{
			prob_improvement(&mode_exc,
				p_alpha_exc, p_beta_exc, p_gamma_exc,
				exp_cnt_exc, error_exc,
				fS, fD,
				alpha_delta, beta_delta, gamma_delta
				//, group
			);
		}

		if ((verbose) && (group == print_group))
		{
			printf("\n\n0 (%d, %d) expected_count_inh %f, expected_count_exc %f, modes %d, %d",
				group, j, exp_cnt_inh, exp_cnt_exc, 
				mode_inh, mode_exc);
			// if ((error_inh > p))
				printf("\n1 (%d, %d) alpha_inh %f, beta_inh %f , gamma_inh %f  \nerror=%f",
					group, j, *p_alpha_inh, *p_beta_inh, *p_gamma_inh,  exp_cnt_inh - fS);
			// if ((error_exc > p))
				printf("\n2 (%d, %d) alpha_exc %f, beta_exc %f , gamma_exc %f  \nerror=%f",
					group, j, *p_alpha_exc, *p_beta_exc, *p_gamma_exc,  exp_cnt_exc - fS);
		}

	}

	return j;
}

__global__ void fill_G_exp_ccsyn_per_src_type_and_delay_(
	const int S,
	const float fS,
	const int D,
	const float fD,
	const int G,
	const int* G_neuron_counts,
	float* G_conn_probs,
	int* G_exp_ccsyn_per_src_type_and_delay,
	bool verbose = 0,
	int print_group = 1
)
{
	// connection probabilities from inh -> exc
	// connection probabilities from exc -> (inh & exc)

	const int g = threadIdx.x + blockDim.x * blockIdx.x;

	if (g < G)
	{
		const int ioffs_inh = 2 * G + g;
		const int ioffs_exc = (2 + D) * G + g;

		float alpha_inh = 2.f;
		float alpha_exc = 1.f;
		float beta_inh = 1.f + fD / 3.f;
		float beta_exc = 1.f;
		float gamma_inh = .01f;
		float gamma_exc = .125f;

		const float alpha_delta = 0.04f;
		const float beta_delta = 0.04f;
		const float gamma_delta = 0.01f;



		const int opt_runs = roughly_optimize_connection_probabilites(
				fS,
				fD, 
				D,
				G,
				G_neuron_counts, 
				ioffs_inh, ioffs_exc,
				&alpha_inh, &beta_inh, &gamma_inh,
				&alpha_exc, &beta_exc, &gamma_exc,
				alpha_delta, beta_delta, gamma_delta, 
				g, 
				verbose);

		if ((g < 10) && (opt_runs > 98) || ((g == print_group) && (verbose))) {
			printf("\n(GPU: optimize_connection_probabilites) group(%d, opt_runs) = %d", g, opt_runs);
		}
		// if ((verbose) && (g == print_group)) {
		// 	printf("\nalpha_inh = %f, beta_inh = %f, gamma_inh = %f", alpha_inh, beta_inh, gamma_inh);
		// 	printf("\nalpha_exc = %f, beta_exc = %f, gamma_exc = %f", alpha_exc, beta_exc, gamma_exc);
		// }

		int expected_synapses_inh = 0;
		int expected_synapses_exc = 0;

		// int delay_with_min_exp_inh_syn_ge1 = 0
		int delay_with_max_inh_targets = 0;
		int exp_inh_syn_with_max_targets = 0;
		int max_inh_targets = 0;
		// int delay_with_min_exp_exc_syn_ge1 = 0
		int delay_with_max_exc_targets = 0;
		int exp_exc_syn_with_max_targets = 0;
		int max_exc_targets = 0;

		int idx = 0;
		int exc_syn_count = 0;
		int inh_syn_count = 0;

		for (int delay = 0; delay < D; delay++)
		{
			const float fdelay = __int2float_rn(delay);
			float prob_inh = sigmoidal_connection_probability(fdelay, fD, alpha_inh, beta_inh, gamma_inh);
			float prob_exc = sigmoidal_connection_probability(fdelay, fD, alpha_exc, beta_exc, gamma_exc);

			G_conn_probs[(g)*D + delay] = prob_inh;
			G_conn_probs[(G * D) + (g * D) + delay] = prob_exc;

			idx = delay * G;
			const int n_inh_targets = G_neuron_counts[idx + ioffs_exc];
			int n_exc_targets = n_inh_targets + G_neuron_counts[idx + ioffs_inh];
			const float f_n_inh_targets = __int2float_rn(n_inh_targets);
			float f_n_exc_targets = __int2float_rn(n_exc_targets);

			if ((delay == 0) && (G_neuron_counts[ioffs_inh - G] > 0))
			{
				// only exc neurons will have a technical probability > 0 to form an autapse
				n_exc_targets-=1;
				f_n_exc_targets-= 1.f;
			}

			inh_syn_count = min(__float2int_ru(prob_inh * f_n_inh_targets), n_inh_targets);
			expected_synapses_inh += inh_syn_count;
			G_exp_ccsyn_per_src_type_and_delay[g + idx + G] = expected_synapses_inh;
			
			idx += (D + 1) * G;
			exc_syn_count = __float2int_ru(prob_exc * f_n_exc_targets);
			expected_synapses_exc += exc_syn_count;
			G_exp_ccsyn_per_src_type_and_delay[g + idx + G] = expected_synapses_exc;

			if ((n_inh_targets > max_inh_targets)){
				exp_inh_syn_with_max_targets = inh_syn_count;
				delay_with_max_inh_targets = delay;
				max_inh_targets = n_inh_targets;
			}
			if ((n_exc_targets > max_exc_targets)){
				exp_exc_syn_with_max_targets = exc_syn_count;
				delay_with_max_exc_targets = delay;
				max_exc_targets = n_exc_targets;
			}

			// expected_synapses_inh += min(__float2int_ru(prob_inh * f_n_inh_targets), n_inh_targets);
			if ((verbose) && (g == print_group)) {
				printf("\nexp inh %f", prob_inh * f_n_inh_targets);
				printf("\nexp exc %f -> %d | %f (sum=%d)", 
					prob_exc * f_n_exc_targets, 
					min(__float2int_ru(prob_exc * f_n_exc_targets), n_exc_targets),
					roundf(prob_exc * f_n_exc_targets + .5),
					expected_synapses_exc
				);  
			}
		}
		
		// int res_inh = G_exp_ccsyn_per_src_type_and_delay[g + idx - (D * G)];
		// int res_exc = G_exp_ccsyn_per_src_type_and_delay[g + idx + G];

		if ((expected_synapses_inh != S)){
			int add = S - expected_synapses_inh;
			if (expected_synapses_inh > S){
				if (exp_inh_syn_with_max_targets < 1)  
				{
					add = 0;
					printf("\n(GPU: optimize_connection_probabilites) delay_inh(g=%d, exp_too_low=%d, max_targets=%d) = %d", 
					       g, exp_inh_syn_with_max_targets, max_inh_targets, delay_with_max_inh_targets);
				}
			} else if (exp_inh_syn_with_max_targets >= max_inh_targets){
				add = 0;
				printf("\n(GPU: optimize_connection_probabilites) delay_inh(g=%d, exp_too_high=%d, max_targets=%d) = %d", 
					   g, exp_inh_syn_with_max_targets, max_inh_targets, delay_with_max_inh_targets);
			}
			if (add != 0){
				for (int delay = delay_with_max_inh_targets; delay < D; delay++){
					G_exp_ccsyn_per_src_type_and_delay[g + (delay + 1) * G] += add;
				}
			}
			// printf("\n(%d) %d -> %d ", g, expected_synapses_inh, G_exp_ccsyn_per_src_type_and_delay[g + idx + G]);
		}

		if (expected_synapses_exc != S){
			int add = S - expected_synapses_exc;
			if (expected_synapses_exc > S){
				if (exp_exc_syn_with_max_targets < 1)  
				{
					add = 0;
					printf("\n(GPU: optimize_connection_probabilites) delay_exc(g=%d, exp_too_low=%d, max_targets=%d) = %d", 
						   g, exp_exc_syn_with_max_targets, max_exc_targets, delay_with_max_exc_targets);
				}
			} else if (exp_exc_syn_with_max_targets >= max_exc_targets){
				add = 0;
				printf("\n(GPU: optimize_connection_probabilites) delay_exc(g=%d, exp_too_high=%d, max_targets=%d) = %d", 
					   g, exp_exc_syn_with_max_targets, max_exc_targets, delay_with_max_exc_targets);
			} 
			if (add != 0){
				for (int delay = delay_with_max_exc_targets; delay < D; delay++){
					G_exp_ccsyn_per_src_type_and_delay[ g + (delay + 2 + D) * G] += add;
				}
			}
			if (G_exp_ccsyn_per_src_type_and_delay[g + (2 * D + 1) * G] != S){
				printf("\n(GPU: optimize_connection_probabilites) add(g=%d, exp=%d, max_targets=%d) = %d (%d, %d)", 
					   g, exp_exc_syn_with_max_targets, max_exc_targets, add, expected_synapses_exc,
					   G_exp_ccsyn_per_src_type_and_delay[g + (2 * D + 1) * G]);
			}

		} 

		if ((verbose) && (g == print_group)) {
			printf("\nres_inh = %d", expected_synapses_inh);
			printf("\nres_exc = %d", expected_synapses_exc);
		}
	}
}



void fill_G_exp_ccsyn_per_src_type_and_delay(
	const int S,
	const int D,
	const int G,
	const int* G_neuron_counts,
	float* G_conn_probs,
	int* G_exp_ccsyn_per_src_type_and_delay,
	bool verbose
)
{	
	hipDeviceSynchronize();
	LaunchParameters launch(G, (void *)fill_G_exp_ccsyn_per_src_type_and_delay_); 

	fill_G_exp_ccsyn_per_src_type_and_delay_ KERNEL_ARGS2(launch.grid3, launch.block3)(
		S,
		static_cast<float>(S),
		D,
		static_cast<float>(D),
		G,
		G_neuron_counts,
		G_conn_probs,
		G_exp_ccsyn_per_src_type_and_delay,
		verbose
	);
	
	hipDeviceSynchronize();
	printf("\n\n");
}


__device__ int relative_typed_delay_rep_index(
	const int N_autapse_idx,
	const int G_rep_idx0,
	const int G_rep_idx1,
	const int g_N_count,
	const int* G_rep,
	const int n_groups,
	const int* cc_snk,
	bool verbose

)
{
	if (g_N_count == 0)
	{
		return -1;
	}

	int G_rep_idx = G_rep_idx0;
	int g = G_rep[G_rep_idx];
	int Ng_start = cc_snk[g];
	const int Ng_last = cc_snk[G_rep[G_rep_idx1] +1];

	
	if ((N_autapse_idx < Ng_start) || (N_autapse_idx >= Ng_last))
	{
		return -1;
		if (verbose)
		{
			printf(
			"(search, not in range) g=(%d), n=%d, G_rep[%d: %d], Ng_start=%d, Ng_last=%d\n", 
			g, N_autapse_idx, G_rep_idx0, G_rep_idx1, Ng_start, Ng_last);
		}
	}

	int result = N_autapse_idx;
	result -= Ng_start;
	int Ng_next = cc_snk[g + 1];

	if (verbose)
	{
		printf("(search) g=(%d), n=%d, Ng_start=%d, Ng_next=%d\n", g, N_autapse_idx, Ng_start, Ng_next);
	}
	//if (bprint)
	//{
	//	printf("\n  search (%d) g_start_col %d, n_g_search %d, g=%d",
	//		N_autapse_idx, g_search_start_col, n_g_search, g );
	//	printf("\n  (%d)  [%d], src_loc %d, g = %d [%d, %d]... %d]",
	//		N_autapse_idx, result, src_loc, g, start_col_next_group, end_col_next_group, last_col);
	//}
	
	bool found = (N_autapse_idx >= Ng_start) && (N_autapse_idx < Ng_next);

	int Ng_prev = Ng_next;
	
	while ((!found) && (G_rep_idx < G_rep_idx1))
	{
		G_rep_idx++;
		
		g = G_rep[G_rep_idx];
		Ng_start = cc_snk[g];
		Ng_next = cc_snk[g + 1];

		result -= (Ng_start - Ng_prev);

		found = (N_autapse_idx >= Ng_start) && (N_autapse_idx < Ng_next);
		if (verbose)
		{
			printf("(search, found=%d) g=(%d), n=%d, Ng_start=%d, Ng_next=%d\n", found, g, N_autapse_idx, Ng_start, Ng_next);
		}
	}
	return result * found + (-1) * (!found);
}


__global__ void fill_relative_autapse_indices_(
	const int D,
	const int G,
	const int* cc_src,
	const int* cc_snk,
	const int* G_rep,
	const int* G_delay_counts,
	int* G_autapse_indices,
	int* G_relative_autapse_indices,
	bool verbose = 0,
	int print_group = 1
)
{
	const int g = blockIdx.x * blockDim.x + threadIdx.x;  // NOLINT(bugprone-narrowing-conversions, cppcoreguidelines-narrowing-conversions)
	
	if (g < G)
	{
		
		const int N_autapse_idx = cc_src[g];
		const int g_N_count = cc_src[g + 1] - N_autapse_idx;
	
		for (int d=0; d < D; d++)
		{
			const int g_rep_col0 = G_delay_counts[g * (D + 1) + d];
			const int g_rep_col1 = G_delay_counts[g * (D + 1) + d+1]-1;
			const int G_rep_idx0 = g * G + g_rep_col0;
			const int G_rep_idx1 = g * G + g_rep_col1;
			const int n_groups = g_rep_col1 - g_rep_col0 +1;

			if (verbose && (g == print_group))
			{
				printf("g=(%d), n=%d, d=%d, g_rep_cols=[%d, %d], idcs=[%d,%d], groups=[%d, ...,%d]\n", 
					g, N_autapse_idx, d, 
					g_rep_col0, g_rep_col1,
					G_rep_idx0, G_rep_idx1,
					G_rep[G_rep_idx0], G_rep[G_rep_idx1]);
			}

			const int relative_autapse_index = relative_typed_delay_rep_index(
				N_autapse_idx,
				G_rep_idx0,
				G_rep_idx1,
				g_N_count,
				G_rep,
				n_groups,
				cc_snk,
				verbose && (g == print_group)
			);

			G_relative_autapse_indices[g + d * G] = relative_autapse_index;

			if (relative_autapse_index != -1){
				G_autapse_indices[g + d * G] = N_autapse_idx;
			} else {
				G_autapse_indices[g + d * G] = -1;
			}

			if (verbose && (g == print_group))
			{
				printf("g=(%d), n=%d, d=%d, N=%d, rN=%d\n", 
					g, N_autapse_idx, d, 
					G_autapse_indices[g + d * G],
					relative_autapse_index);
			}

			//if (bprint)
			//{
			//	printf("\nres: %d << %d, g = %d, d = %d\n---------\n", relative_self_index[g + d * G], self_index[g + d * G],g,d);
			//}

		}
	}
}

__forceinline__ __device__ int random_uniform_int(hiprandState *local_state, const float min, const float max)
{
	return __float2int_rd(fminf(min + hiprand_uniform(local_state) * (max - min + 1.f), max));
}

__device__ int random_uniform_int_with_exclusion(
	hiprandState *local_state, 
	const float minf, 
	const float maxf,
	const float maxf0,
	const bool exclude,
	const int autapse_idx,
	const int n, 
	const int s
){
	int new_sink = __float2int_rd(fminf(minf + hiprand_uniform(local_state) * (maxf - minf + 1.f), maxf));
	int i = 0;
	if (exclude){
		while ((new_sink==autapse_idx) && (i<50))
		{
			new_sink = __float2int_rd(fminf(minf + hiprand_uniform(local_state) * (maxf - minf + 1.f), maxf));
			i++;
		}

		if (i < 50){ return new_sink; } else { 
			printf("\n Loop-Warning [autapse_2](%d, %d) range=[%f, %f] -> [%f, %f], autapse_idx=%d, sink=%d", 
				   n, s, 0.f, maxf0, minf, maxf, autapse_idx, new_sink); 
		}
	}
}


__device__ void print_array(int* arr, int r, int c){
	
	printf("\n\n");
	for (int j=0; j < r; j++)
	{
		for (int i=0; i < c; i++)
		{
			printf("%d ", arr[i + j * c]);
		}
		printf("\n");
	}	
}


__device__ bool duplicated_int(
	const int min_hit, const int max_hit,
	const int row_idx0,
	const int n, const int s, 
	const float maxf0, const float minf, const float maxf, 
	const int delay_col0, const int delay_col1, const int autapse_idx, const int new_sink,
	int* N_rep,
	const int k
){
	int i = delay_col0;

	if ((k==45)){
		printf("\n Loop-Warning [duplicated](%d, %d) range=[%f, %f] -> [%f, %f], delay_cols=[%d,%d], autapse_idx=%d, sink=%d", 
			n, s, 0.f, maxf0, minf, maxf, delay_col0, delay_col1, autapse_idx, new_sink);}
	
	// check if the drawn integer has alredy been set

	if ((min_hit < new_sink) && (new_sink < max_hit)) {
		while (i < s) {
			if (N_rep[row_idx0 + i] == new_sink) { 
				if ((k >= 45)){
					printf("\nLoop-Warning [duplicated (%d)](%d, %d), col0=%d, write_idx=%d, k=%d) rep=%d sink=%d", 
					true, n, s, i, row_idx0 + i, k, N_rep[row_idx0 + i], new_sink);}
				return true; }
			i++;
		}
	} else if ((new_sink == min_hit) || (new_sink == max_hit)){ 
		if ((k >= 45)){
			printf("\nLoop-Warning [duplicated (%d) hit](%d, %d), col0=%d, write_idx=%d, k=%d) rep=%d sink=%d", 
			true, n, s, i, row_idx0 + i, k, N_rep[row_idx0 + i], new_sink);}
		return true; }
	return false;
}


__global__ void k_set_locally_indexed_connections(
	const int N,
	const int S,
	const int D,
	const int G,
	hiprandState* curand_states,
	const int* N_G,
	const int* cc_src,
	const int* G_neuron_counts,
	const int* G_relative_autapse_indices,
	bool has_autapses,
	const int gc_location0,
	const int gc_location1,
	const int gc_conn_shape0,
	const int gc_conn_shape1,
	//const float init_weight,
	//float* weights,
	int* N_delays,
	const int* cc_syn,
	int* sort_keys,
	int* N_rep,
	bool verbose
)
{
	extern __shared__ int sh_delays[];
	int* n_targets = &sh_delays[(D+1) * blockDim.x];

	const int n = gc_location0 + blockIdx.x * blockDim.x + threadIdx.x;
	
	if (n < gc_location0 + gc_conn_shape0)
	{
		hiprandState local_state = curand_states[n];
		
		const int src_loc = N_G[n * 2 + 1];
		int tdx = threadIdx.x;
		const int row_idx0 = n * S;

		sh_delays[tdx] = 0;
		N_delays[n] = 0;

		// if (n == gc_location0){print_array(sh_delays, 2 * D + 1, blockDim.x);}

		for (int d=1; d<D+1; d++)
		{
			int end_rep_col = cc_syn[src_loc + d * G];
			
			sh_delays[tdx + d * blockDim.x] = gc_location1 + end_rep_col;
			n_targets[tdx + (d-1)* blockDim.x] = G_neuron_counts[src_loc + (d-1) * G];
			
			N_delays[n + d * N] += end_rep_col;
			
		}

		if ((verbose) && (n == gc_location0)){ print_array(sh_delays, 2 * D + 1, blockDim.x); }

		int sort_key = row_idx0; // + gc_location1 + max(0, (D - S) * n);
		
		// [delay_col0, delay_col1]: column-interval in which to write sink neurons
		int delay = 0;
		int delay_col0 = sh_delays[tdx];
		int delay_col1 = sh_delays[tdx + blockDim.x];
		int n_rep_cols = delay_col1 - delay_col0;
		
		// [min, max]: interval from which to draw an integer ('sink'-neuron)
		int min = 0;
		int max = n_targets[tdx] - 1;
		float maxf = __int2float_rn(max);
		float minf = 0.f;
		float maxf0 = maxf;
		
		int autapse_idx = -1;
		if (has_autapses){ 
			autapse_idx = G_relative_autapse_indices[src_loc + delay * G] + (n - cc_src[src_loc]); 
		}
		int new_sink;

		int min_hit = -1;
		int max_hit = gc_location1 + gc_conn_shape1;
				
		// fill N_rep[n, s] for in [gc_location1, gc_location1 + gc_conn_shape1]
		for (int s = gc_location1; s < gc_location1 + gc_conn_shape1; s++)
		{
			const int write_idx = row_idx0 + s;

			while ((s == delay_col1) && (delay < D+1))
			{
				// if we reach the end of the write interval, update all variables
				if (delay >= 1){ autapse_idx = -1; }

				tdx += blockDim.x;
				delay_col0 = sh_delays[tdx];
				delay_col1 = sh_delays[tdx + blockDim.x];
				n_rep_cols = delay_col1 - delay_col0;
				if (n_rep_cols >0)
				{
					min_hit = -1;
					max_hit = gc_location1 + gc_conn_shape1;
					min = 0;
					max = n_targets[tdx] - 1;
					minf = 0.f;
					maxf0 = __int2float_rn(max);
					maxf = maxf0;
					sort_key = write_idx;				
				}
				delay++;
			}

			if (n_rep_cols > 0) {	

				if (min > max){ printf("\n Warning [min>max] (%d, %d) %d > %d, range=[%f, %f]", n, s, min, max, 0.f, maxf0); }

				new_sink = random_uniform_int_with_exclusion(&local_state, minf, maxf, maxf0, (has_autapses) && (delay == 0), autapse_idx, n, s);

				
				if (s == delay_col0){
					min_hit = new_sink;
					max_hit = new_sink;
				} else {
					
					int k = 0;
					bool duplicated = true;
					while (duplicated && (k<=50))
					{
						duplicated = duplicated_int(min_hit, max_hit, row_idx0, n, s, maxf0, minf, maxf, delay_col0, delay_col1, autapse_idx, new_sink, &N_rep[0], k);
						if (duplicated) {
							new_sink = random_uniform_int_with_exclusion(&local_state, minf, maxf, maxf0, (has_autapses) && (delay == 0), autapse_idx, n, s);
						}
						k++;
					}
				} 

				// we can narrow the range if we hit the border
				if (new_sink > max){ printf("\n Loop-Warning [new_sink>max] (%d, %d) range=[%f, %f] -> [*, %d], sink=%d", n, s, 0.f, maxf0, max, new_sink); }
				else if (new_sink == max){ max--; maxf -= 1.f; }
				else if (new_sink == min){ min++; minf += 1.f; }
				else if (new_sink < min_hit){ min_hit = new_sink; }
				else if (new_sink > max_hit){ max_hit = new_sink; }
				
				sort_keys[write_idx] = sort_key;
				N_rep[write_idx] = new_sink;	
			}	
		}
		
		curand_states[n] = local_state;
	}
}

void fill_N_rep(
	const int N,
	const int S,
	const int D,
	const int G,
	hiprandState* curand_states,
	const int n_curand_states,
	const int* N_G,
	const int* cc_src,
	const int* cc_snk,
	const int* G_rep,
	const int* G_neuron_counts,
	const int* G_group_delay_counts,
	int* G_autapse_indices,
	int* G_relative_autapse_indices,
	const bool has_autapses,
	const int gc_location0,
	const int gc_location1,
	const int gc_conn_shape0,
	const int gc_conn_shape1,
	const int* cc_syn,
	int* N_delays,
	int* sort_keys,
	int* N_rep,
	bool verbose
)
{
	printf("\nConnecting: ((%d, %d), (%d, %d))", gc_location0, gc_location1, gc_conn_shape0, gc_conn_shape1);
	hipDeviceSynchronize();
	LaunchParameters launch(G, (void *)fill_relative_autapse_indices_); 
	fill_relative_autapse_indices_ KERNEL_ARGS2(launch.grid3, launch.block3)(
		D,
		G,
		cc_src,
		cc_snk,
		G_rep,
		G_group_delay_counts,
		G_autapse_indices,
		G_relative_autapse_indices,
		false);
	hipDeviceSynchronize();

	LaunchParameters l(gc_conn_shape0, (void*)k_set_locally_indexed_connections);
	hipDeviceSynchronize();
	
	k_set_locally_indexed_connections KERNEL_ARGS3(l.grid3, l.block3, l.block3.x * ((2 * D) + 1) * sizeof(int))(
		N,
		S,
		D,
		G,
		curand_states,
		N_G,
		cc_src,
		G_neuron_counts,
		G_relative_autapse_indices,
		has_autapses,
		gc_location0,
		gc_location1,
		gc_conn_shape0,
		gc_conn_shape1,
		//group_conn.initial_weight,
		//weights,
		N_delays,
		cc_syn,
		sort_keys,
		N_rep,
		verbose
	  );

	  printf("\n");
}